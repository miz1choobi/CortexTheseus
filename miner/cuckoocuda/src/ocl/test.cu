#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include "hip/hip_runtime_api.h"
#include "hip/hip_runtime.h"

#include "hip/device_functions.h"
#define n 4

__device__ uint64_t toLong(uint2 a){
	uint64_t l = (uint64_t)a.y << 32 | (uint64_t)a.x;
	return l;
}

__global__ void kernel(uint2 *a, ulonglong4 *b){
	uint2 tmp = {1UL << 31, 1UL << 0};
	for (uint32_t i = 0 ; i < sizeof(uint32_t) * 2; i++ ) {
		printf("%lx ", *((uint8_t*)(((char*)&tmp) + i)));
	}
	printf("\n");
	printf("x  = %x %x\n", tmp.x, tmp.y);
	printf("lx = %lx %lx\n", tmp.x, tmp.y);
	printf("\n");
	printf("1: %lu %lu\n", a[0].x, a[0].y);
	printf("2: %lu %lu\n", a[1].x, a[1].y);
	printf("3: %lu %lu\n", a[2].x, a[2].y);
	printf("4: %lu %lu\n", a[3].x, a[3].y);
	printf("\n");
	printf("1: %u %u\n", a[0].x, a[0].y);
	printf("2: %u %u\n", a[1].x, a[1].y);
	printf("3: %u %u\n", a[2].x, a[2].y);
	printf("4: %u %u\n", a[3].x, a[3].y);
	*b = *(ulonglong4*)(&a[0]);
	printf("%lu %lu %lu %lu\n", toLong(a[0]), toLong(a[1]), toLong(a[2]), toLong(a[3]));

	printf("%llu %llu %llu %llu\n",b->x, b->y, b->z, b->w);

	int xx = 0x1234;
	char cxx = *(char*)&xx;
	if(cxx == 0x12){
		printf("big\n");
	}else{
		printf("little\n");
	}
}

__global__ void kernel2(ulonglong4 *a){
	printf("%lu %lu %lu %lu\n", a[0].x, a[0].y, a[0].z, a[0].w);
}

int main(){
	uint32_t *a = (uint32_t*)malloc(sizeof(uint32_t) * n * 2);
	ulonglong4 b;
	int *deva, *devb;
	hipMalloc((void**)&deva, sizeof(uint2)*n);
	hipMalloc((void**)&devb, sizeof(uint2)*n);	
	for(int i = 0; i < 8; i++){
		a[i] = (uint32_t)(i);
//		printf("%llu ", a[i]);
	}
//	printf("\n");
//	printf("%llu\n", sizeof(uint2));
	hipMemcpy(deva, a, sizeof(uint2)*n, hipMemcpyHostToDevice);
//	kernel<<<1,1>>>((uint2*)deva, (ulonglong4*)devb);
	kernel2<<<1,1>>>((ulonglong4*)deva);
	hipMemcpy(&b, devb, sizeof(ulonglong4), hipMemcpyDeviceToHost);
//	printf("%llu %llu %llu %llu\n", b.x, b.y, b.z, b.w);
	return 0;
}
